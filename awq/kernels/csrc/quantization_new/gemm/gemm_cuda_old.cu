#include "hip/hip_runtime.h"
#include <hip/hip_fp16.h>
#include "semaphore.h"
#include "gemm_cuda.h"
#include "../dequantize.cuh"
#include <torch/extension.h>
#include <cuda_pipeline_primitives.h>

#define kInterleave 4
#define OP_M 16
#define OP_N 8
#define OP_K 16
#define INTRIN_M 16
#define INTRIN_N 16
#define INTRIN_K 16
#define WARP_SIZE 32
#define SMEM_PAD_A 0
#define SMEM_PAD_B 0
#define PACK_SIZE 8
#if (__CUDACC_VER_MAJOR__ >= 11) && (__CUDACC_VER_MINOR__ >= 4)
#define L2_CACHEHINT(size) ".L2::" #size "B"
#else
#define L2_CACHEHINT(size)
#endif

#define KERNEL_LAUNCH_CODE                                                                                                                              \
  int num_mn_tiles = (num_in_feats + CTA_M - 1) / CTA_M * (num_out_channels + CTA_N - 1) / CTA_N;                                                       \
  torch::Tensor _semaphores = torch::empty({num_mn_tiles}, options_int);                                                                                \
  auto semaphores = reinterpret_cast<int *>(_semaphores.data_ptr<int>());                                                                               \
  constexpr int NUM_WARPS = (CTA_M / WARP_M) * (CTA_N / WARP_N) * (CTA_K / WARP_K);                                                                     \
  constexpr int SCALES_SMEM_SIZE = (G >= CTA_K) ? (CTA_N / (G / CTA_K) * STAGES * 2) : (CTA_N * (CTA_K / G) * STAGES * 2);                              \
  constexpr int kSmemByteSize = (CTA_M * (CTA_K + SMEM_PAD_A) + CTA_N * (CTA_K + SMEM_PAD_B) / kInterleave + SCALES_SMEM_SIZE) * STAGES * sizeof(half); \
  if (kSmemByteSize >= 99 * 1024)                                                                                                                       \
  {                                                                                                                                                     \
    printf("This kernel requires %d Bytes of shared memory, which exceeds device limit.\n", kSmemByteSize);                                             \
    return _out_feats;                                                                                                                                  \
  }                                                                                                                                                     \
  int j_factors1 = num_out_channels / CTA_N / 1;                                                                                                        \
  dim3 num_blocks((num_out_feats + CTA_M - 1) / CTA_M * j_factors1 * SPLITK);                                                                           \
  dim3 threads_per_block(WARP_SIZE, NUM_WARPS);                                                                                                         \
  auto kernel_func = gemm_w4a16_T1<CTA_M, CTA_N, CTA_K, WARP_M, WARP_N, WARP_K, STAGES, G, SPLITK>;                                                     \
  hipFuncSetAttribute(reinterpret_cast<const void*>(kernel_func), hipFuncAttributeMaxDynamicSharedMemorySize, kSmemByteSize);                                                        \
  kernel_func<<<num_blocks, threads_per_block, kSmemByteSize>>>(                                                                                        \
      in_feats, kernel, scales, zeros, out_feats, semaphores, num_in_feats, num_out_channels, num_in_channels);

template <int N>
__inline__ __host__ __device__ int get_log_tile(int n)
{
  if (N >= 8 && n >= 6)
    return 3;
  else if (N >= 4 && n >= 3)
    return 2;
  else if (N >= 2 && n >= 2)
    return 1;
  else
    return 0;
}

__inline__ __device__ uint2 get_block_idx_mapping(int blockIdx_x, int blockIdx_y, int log_tile)
{
  return make_uint2((blockIdx_x >> log_tile), (blockIdx_y << log_tile) + ((blockIdx_x) & ((1 << (log_tile)) - 1)));
}

template <int SLICES, int NUM_WARPS_MN>
__device__ void sync_slice(int slice_id)
{
  if constexpr (SLICES == 1)
  {
    __syncthreads();
  }
  else
  {
    constexpr int SLICE_GROUP = (SLICES + 7) / 8;
    constexpr uint32_t num_threads = NUM_WARPS_MN * WARP_SIZE;
    const uint32_t barrier_id = slice_id / SLICE_GROUP + 1;
    asm volatile("bar.sync %0, %1;" : : "r"(barrier_id), "n"(num_threads));
  }
}

__inline__ __device__ uint32_t cast_smem_ptr_to_uint(void const *const ptr)
{
  uint32_t smem_int_ptr;

  asm("{.reg .u64 smem_ptr; cvta.to.shared.u64 smem_ptr, %1; cvt.u32.u64 %0, smem_ptr; }\n"
      : "=r"(smem_int_ptr)
      : "l"(ptr));

  return smem_int_ptr;
}

__inline__ __device__ void ldmatrix_m8n8_x4_b16(half *shared_warp, int ax0_0, uint32_t addr)
{
  __asm__ __volatile__(
      "ldmatrix.sync.aligned.m8n8.x4.shared.b16"
      "{%0, %1, %2, %3}, [%4];"
      : "=r"(((unsigned *)(shared_warp + (ax0_0 * 8)))[0]), "=r"(((unsigned *)(shared_warp + (ax0_0 * 8)))[1]), "=r"(((unsigned *)(shared_warp + (ax0_0 * 8)))[2]), "=r"(((unsigned *)(shared_warp + (ax0_0 * 8)))[3])
      : "r"(addr));
}

__inline__ __device__ void ldmatrix_m8n8_x4_trans_b16(half *shared_warp, int ax0_0, uint32_t addr)
{
  __asm__ __volatile__(
      "ldmatrix.sync.aligned.m8n8.x4.trans.shared.b16"
      "{%0, %1, %2, %3}, [%4];"
      : "=r"(((unsigned *)(shared_warp + (ax0_0 * 8)))[0]), "=r"(((unsigned *)(shared_warp + (ax0_0 * 8)))[1]), "=r"(((unsigned *)(shared_warp + (ax0_0 * 8)))[2]), "=r"(((unsigned *)(shared_warp + (ax0_0 * 8)))[3])
      : "r"(addr));
}

__inline__ __device__ void cp_async_cg_A(uint32_t smem_int_ptr, const uint4 *__restrict__ src, bool mask)
{
  const int cp_size = 16;
  asm volatile("{"
               "  .reg .pred p;"
               "  setp.ne.b32 p, %0, 0;"
               "  @p cp.async.cg.shared.global" L2_CACHEHINT(128) " [%1], [%2], %3;"
                                                                  "}" ::"r"((int)mask),
               "r"(smem_int_ptr),
               "l"(src),
               "n"(cp_size));
}

__device__ __inline__ void mma_m16n8k16(float *C_warp, half *A_shared_warp, half *B_shared_warp)
{
  __asm__ __volatile__(
      "mma.sync.aligned.m16n8k16.row.col.f32.f16.f16.f32"
      "{%0, %1, %2, %3}, {%4, %5, %6, %7}, {%8, %9}, {%10, %11, %12, %13};"
      : "=f"(((float *)C_warp)[0]), "=f"(((float *)C_warp)[1]), "=f"(((float *)C_warp)[2]), "=f"(((float *)C_warp)[3])
      : "r"(((unsigned *)A_shared_warp)[0]), "r"(((unsigned *)A_shared_warp)[1]), "r"(((unsigned *)A_shared_warp)[2]), "r"(((unsigned *)A_shared_warp)[3]), "r"(((unsigned *)B_shared_warp)[0]), "r"(((unsigned *)B_shared_warp)[1]), "f"(((float *)C_warp)[0]), "f"(((float *)C_warp)[1]), "f"(((float *)C_warp)[2]), "f"(((float *)C_warp)[3]));
}

template <int CTA_M, int CTA_N, int CTA_K, int CTA_SIZE, int SHARED_K_ITERS, int STAGES>
__device__ __inline__ void global_to_share_one_stage_A(half *src, half *dst, int global_nrows, int global_ncols, int cta_offset_m, int cta_offset_n, int cta_offset_k, int global_iter_k, int shared_iter_k, bool mask)
{
  constexpr int threads_needed = (CTA_M * CTA_K) / PACK_SIZE / SHARED_K_ITERS;
  constexpr int threads_used = threads_needed < CTA_SIZE ? threads_needed : CTA_SIZE;
  constexpr int total_global_iters = (CTA_M * CTA_K) / PACK_SIZE / threads_used;
  constexpr int partial_global_iters = (total_global_iters + SHARED_K_ITERS - 1) / SHARED_K_ITERS;
  constexpr int cta_step_m_or_n = (threads_used * PACK_SIZE) / CTA_K;
  constexpr int warp_step_m_or_n = (WARP_SIZE * PACK_SIZE) / CTA_K;
  constexpr int threads_per_row = CTA_K / PACK_SIZE;
  constexpr int kSmemCol = CTA_K + SMEM_PAD_A;
  bool local_mask = mask & (threadIdx.y * WARP_SIZE + threadIdx.x < threads_used);
  int ld_col = (threadIdx.x % threads_per_row);
#pragma unroll
  for (int _global_iter = 0; _global_iter < partial_global_iters; ++_global_iter)
  {
    int global_iter = shared_iter_k * partial_global_iters + _global_iter;
    int ld_row = global_iter * cta_step_m_or_n + threadIdx.y * warp_step_m_or_n + (threadIdx.x / threads_per_row);
    int ld_col_swizzled = (ld_col ^ (ld_row) & 7) * PACK_SIZE;
    void *dst_ptr = (void *)(dst + ld_row * kSmemCol + ld_col_swizzled);
    uint4 *src_ptr = (uint4 *)(src + (ld_row + cta_offset_m) * global_ncols + ld_col * PACK_SIZE + global_iter_k * CTA_K + cta_offset_k); // cta_offset_m * global_ncols + global_iter * cta_step_m_or_n * global_ncols + threadIdx.y * warp_step_m_or_n * global_ncols + (threadIdx.x / threads_per_row) * global_ncols + global_iter_k * CTA_K + (threadIdx.x % threads_per_row) * PACK_SIZE);
    if constexpr (STAGES > 1)
    {
      uint32_t addr = cast_smem_ptr_to_uint(dst_ptr);
      cp_async_cg_A(addr, src_ptr, local_mask & (ld_row + cta_offset_m < global_nrows));
    }
    else
    {
      if (local_mask & (ld_row + cta_offset_m < global_nrows))
        *(uint4 *)dst_ptr = *src_ptr;
    }
  }
}

template <int CTA_M, int CTA_N, int CTA_K, int CTA_SIZE, int SHARED_K_ITERS, int STAGES>
__device__ __inline__ void global_to_share_one_stage_B(half *src, half *dst, int global_ncols, int cta_offset_m, int cta_offset_n, int cta_offset_k, int global_iter_k, int shared_iter_k, bool mask)
{
  constexpr int threads_needed = (CTA_N / kInterleave * CTA_K) / PACK_SIZE / SHARED_K_ITERS;
  constexpr int threads_used = threads_needed < CTA_SIZE ? threads_needed : CTA_SIZE;
  constexpr int total_global_iters = (CTA_N / kInterleave * CTA_K) / PACK_SIZE / threads_used;
  constexpr int partial_global_iters = (total_global_iters + SHARED_K_ITERS - 1) / SHARED_K_ITERS;
  constexpr int cta_step_m_or_n = (threads_used * PACK_SIZE) / CTA_K;
  constexpr int warp_step_m_or_n = (WARP_SIZE * PACK_SIZE) / CTA_K;
  constexpr int threads_per_row = CTA_K / PACK_SIZE;
  constexpr int kSmemCol = CTA_K + SMEM_PAD_B;
  bool local_mask = mask & (threadIdx.y * WARP_SIZE + threadIdx.x < threads_used);
#pragma unroll
  for (int _global_iter = 0; _global_iter < partial_global_iters; ++_global_iter)
  {
    int global_iter = shared_iter_k * partial_global_iters + _global_iter;

    int ld_row = global_iter * cta_step_m_or_n + threadIdx.y * warp_step_m_or_n + (threadIdx.x / threads_per_row);
    int ld_col = (threadIdx.x % threads_per_row);
    int ld_col_swizzled = ld_col ^ (ld_row % 2) & 7;
    void *dst_ptr = (void *)(dst + (ld_row * kSmemCol + ld_col_swizzled * PACK_SIZE));
    uint4 *src_ptr = (uint4 *)(src + global_iter_k * CTA_K + cta_offset_n / kInterleave * global_ncols + ld_row * global_ncols + ld_col * PACK_SIZE + cta_offset_k);
    if constexpr (STAGES > 1)
    {
      uint32_t addr = cast_smem_ptr_to_uint(dst_ptr);
      cp_async_cg_A(addr, src_ptr, local_mask);
    }
    else
    {
      if (local_mask)
        *(uint4 *)dst_ptr = *src_ptr;
    }
  }
}

template <int CTA_M, int CTA_N, int CTA_K, int CTA_SIZE, int STAGES, int G>
__device__ __inline__ void global_to_share_one_stage_scales(half *src, half *dst, half *src_z, half *dst_z, int global_ncols, int cta_offset_m, int cta_offset_n, int cta_offset_k, int global_iter_k, int shared_iter_k, bool mask)
{
  constexpr int LD_AMOUNT = (G >= CTA_K) ? CTA_N : CTA_N * CTA_K / G;
  constexpr int threads_needed = LD_AMOUNT / PACK_SIZE / 1;
  constexpr int threads_used = threads_needed < CTA_SIZE ? threads_needed : CTA_SIZE;
  constexpr int total_global_iters = LD_AMOUNT / PACK_SIZE / threads_used;
  constexpr int threads_per_row = CTA_N / PACK_SIZE;
  constexpr int kSmemCol = CTA_N;
  bool local_mask = mask & (threadIdx.y * WARP_SIZE + threadIdx.x < threads_used);
  int g_idx = (cta_offset_k + global_iter_k * CTA_K) / G;

  void *dst_ptr = (void *)(dst + (threadIdx.x / threads_per_row) * kSmemCol + (threadIdx.x % threads_per_row) * PACK_SIZE);
  uint4 *src_ptr = (uint4 *)(src + g_idx * global_ncols + cta_offset_n + (threadIdx.x / threads_per_row) * global_ncols + (threadIdx.x % threads_per_row) * PACK_SIZE);
  void *dst_ptr_z = (void *)(dst_z + (threadIdx.x / threads_per_row) * kSmemCol + (threadIdx.x % threads_per_row) * PACK_SIZE);
  uint4 *src_ptr_z = (uint4 *)(src_z + g_idx * global_ncols + cta_offset_n + (threadIdx.x / threads_per_row) * global_ncols + (threadIdx.x % threads_per_row) * PACK_SIZE);
  if (STAGES > 1)
  {
    uint32_t addr = cast_smem_ptr_to_uint(dst_ptr);
    cp_async_cg_A(addr, src_ptr, local_mask);
    uint32_t addr_z = cast_smem_ptr_to_uint(dst_ptr_z);
    cp_async_cg_A(addr_z, src_ptr_z, local_mask);
  }
  else
  {
    if (local_mask)
    {
      *(uint4 *)dst_ptr = *src_ptr;
      *(uint4 *)dst_ptr_z = *src_ptr_z;
    }
  }
}

template <int CTA_M, int CTA_N, int CTA_K, int STAGES, int shared_iters>
__device__ __inline__ void share_to_reg_one_stage_A(half *src, half *dst, int warp_offset_m, int warp_offset_n, int warp_offset_k, int k_0_1)
{
  constexpr int kSmemCol = CTA_K + SMEM_PAD_A;

  for (int shared_iter = 0; shared_iter < shared_iters; ++shared_iter)
  {

    int ld_row = warp_offset_m + shared_iter * OP_M + (threadIdx.x % 16);
    int ld_col = k_0_1 * 16 + (threadIdx.x / 16) * 8 + warp_offset_k;
    int ld_col_swizzled = ((ld_col / PACK_SIZE) ^ (ld_row) & 7) * PACK_SIZE;
    void *addr_ptr = (void *)(src + ld_row * kSmemCol + ld_col_swizzled);

    uint32_t addr = cast_smem_ptr_to_uint(addr_ptr);
    ldmatrix_m8n8_x4_b16(dst, shared_iter, addr);
  }
}

template <int CTA_M, int CTA_N, int CTA_K, int STAGES, bool ldmatrix, int shared_iters, int G>
__device__ __inline__ void share_to_reg_one_stage_B(half *src, half *src_scales, half *src_zeros, half *dst, half *dst_fp16, int warp_offset_m, int warp_offset_n, int warp_offset_k, int k_0_1)
{
  constexpr int kSmemCol = CTA_K + SMEM_PAD_B;
  int r0 = ((threadIdx.x / 8 / 2) * 8 + threadIdx.x % 8);
  int c0 = ((threadIdx.x / 8) % 2) * 8;
  int r = r0 / 4;
  int c = (r0 % 4) * 16 + c0;
  int c_swizzled = ((c / PACK_SIZE) ^ (r % 2) & 7) * PACK_SIZE;

  if constexpr (ldmatrix)
  {
#pragma unroll
    for (int shared_iter = 0; shared_iter < shared_iters; ++shared_iter)
    {
      void *addr_ptr = (void *)(src + warp_offset_n / kInterleave * kSmemCol + shared_iter * 16 / kInterleave * kSmemCol + k_0_1 * 16 + r * kSmemCol + c_swizzled + warp_offset_k);
      uint32_t addr = cast_smem_ptr_to_uint(addr_ptr);
      ldmatrix_m8n8_x4_b16(dst, shared_iter, addr);
    }
  }

#pragma unroll
  for (int shared_iter = 0; shared_iter < shared_iters; ++shared_iter)
  {
    half scale = src_scales[(warp_offset_k / G) * CTA_N + warp_offset_n + 16 * shared_iter + 8 * (k_0_1 % 2) + threadIdx.x / 4];
    half zero = src_zeros[(warp_offset_k / G) * CTA_N + warp_offset_n + 16 * shared_iter + 8 * (k_0_1 % 2) + threadIdx.x / 4];
    half2 scale2 = make_half2(scale, scale);
    half2 zero2 = make_half2(zero, zero);
    half2 loaded[4];

    dequantize_s4_to_fp16x2(*reinterpret_cast<half2 *>(dst + (k_0_1 % 2) * 4 + (k_0_1 / 2 * 2) + shared_iter * 8), reinterpret_cast<uint4 *>(loaded));
#pragma unroll
    for (int i = 0; i < 4; i++)
    {
      loaded[i] = __hfma2(loaded[i], scale2, zero2);
    }
    *reinterpret_cast<uint4 *>(dst_fp16 + shared_iter * 16 + 8 * (k_0_1 % 2)) = *reinterpret_cast<uint4 *>(loaded);
  }
}

template <int CTA_M, int CTA_N, int CTA_K, int WARP_M, int WARP_N, int WARP_K, int STAGES, int G, int SPLITK>
__global__ void gemm_w4a16_T1(half *__restrict__ A, half *__restrict__ B, half *__restrict__ scales, half *__restrict__ zeros, half *__restrict__ C, int *__restrict__ semaphores, int M, int N, int K)
{
  constexpr int NUM_WARPS_MN = CTA_M / WARP_M * CTA_N / WARP_N;
  constexpr int NUM_WARPS = NUM_WARPS_MN * CTA_K / WARP_K;
  constexpr int CTA_SIZE = NUM_WARPS * WARP_SIZE;
  constexpr int CTA_SIZE_MN = NUM_WARPS_MN * WARP_SIZE;
  constexpr int SLICES = CTA_K / WARP_K;
  int num_blocks_n = (N + CTA_N - 1) / CTA_N;
  int num_blocks_m = (M + CTA_M - 1) / CTA_M;
  int blockIdx_x = 0;
  int blockIdx_y = blockIdx.x % (num_blocks_m * num_blocks_n);
  int blockIdx_z = blockIdx.x / (num_blocks_m * num_blocks_n);
  const int log_tile = get_log_tile<1>((N + CTA_N - 1) / CTA_N);
  int blockIdx_m = blockIdx_y / (num_blocks_n >> log_tile);
  int blockIdx_n = blockIdx_y % (num_blocks_n >> log_tile);
  const uint2 block_idx_mapping = get_block_idx_mapping(blockIdx_m, blockIdx_n, log_tile);
  blockIdx_m = block_idx_mapping.x;
  blockIdx_n = block_idx_mapping.y;

  float C_warp[CTA_M * CTA_N / CTA_SIZE_MN];
  constexpr int kSmemPadKA = CTA_K + SMEM_PAD_A;
  constexpr int kSmemPadKB = CTA_K + SMEM_PAD_B;
  constexpr int kSmemSizeAPerStage = CTA_M * kSmemPadKA;
  constexpr int kSmemSizeBPerStage = CTA_N / kInterleave * kSmemPadKB;
  constexpr int kSmemSizeA = kSmemSizeAPerStage * STAGES;
  constexpr int kSmemSizeB = kSmemSizeBPerStage * STAGES;
  constexpr int scales_load_interval = G >= CTA_K ? G / CTA_K : 1;
  constexpr int scales_per_load = G < CTA_K ? CTA_K / G : 1;
  constexpr int kSmemSizeScales = CTA_N * STAGES / scales_load_interval * scales_per_load;
  constexpr int kSmemSizeZeros = CTA_N * STAGES / scales_load_interval * scales_per_load;
  extern __shared__ half mem_shared[];
  half *A_shared = mem_shared;
  half *B_shared = mem_shared + kSmemSizeA;
  half *scales_shared = mem_shared + kSmemSizeA + kSmemSizeB;
  half *zeros_shared = mem_shared + kSmemSizeA + kSmemSizeB + kSmemSizeScales;
  float *C_shared = reinterpret_cast<float *>(mem_shared);
  half A_shared_warp_[2][WARP_M * INTRIN_K /
                         WARP_SIZE];
  half B_shared_warp_[2][WARP_N * 32 /
                         WARP_SIZE];
  half B_shared_warp_tmp_[2][WARP_N * 16 /
                             WARP_SIZE];
  int cta_offset_m = blockIdx_m * CTA_M;
  int cta_offset_n = blockIdx_n * CTA_N;
  int cta_offset_k = blockIdx_z * (K / SPLITK);
  int warp_mn = threadIdx.y % NUM_WARPS_MN;
  int slice_id = threadIdx.y / NUM_WARPS_MN;
  int warp_offset_n = (warp_mn % (CTA_N / WARP_N)) * WARP_N;
  int warp_offset_m = (warp_mn / (CTA_N / WARP_N)) * WARP_M;
  int warp_offset_k = slice_id * WARP_K;

  for (int i = 0; i < CTA_M * CTA_N / CTA_SIZE_MN; i++)
    C_warp[i] = 0.0;

  int gemm_iters = (K + CTA_K - 1) / CTA_K / SPLITK;
  int k_0_0_ld = 0;
  int k_0_0 = 0;
  constexpr int prologue_stages = STAGES == 1 ? 1 : STAGES - 1;
#pragma unroll
  for (k_0_0_ld = 0; k_0_0_ld < prologue_stages; ++k_0_0_ld)
  {
    global_to_share_one_stage_A<CTA_M, CTA_N, CTA_K, CTA_SIZE, 1, STAGES>(A, A_shared + k_0_0_ld * kSmemSizeAPerStage, M, K, cta_offset_m, cta_offset_n, cta_offset_k, k_0_0_ld, 0, true);
    global_to_share_one_stage_B<CTA_M, CTA_N, CTA_K, CTA_SIZE, 1, STAGES>(B, B_shared + k_0_0_ld * kSmemSizeBPerStage, K, cta_offset_m, cta_offset_n, cta_offset_k, k_0_0_ld, 0, true);
    global_to_share_one_stage_scales<CTA_M, CTA_N, CTA_K, CTA_SIZE, STAGES, G>(
        scales, scales_shared + (k_0_0_ld / scales_load_interval * scales_per_load) * CTA_N,
        zeros, zeros_shared + (k_0_0_ld / scales_load_interval * scales_per_load) * CTA_N,
        N, cta_offset_m, cta_offset_n, cta_offset_k,
        k_0_0_ld, 0, k_0_0_ld < gemm_iters && k_0_0_ld % scales_load_interval == 0);
    if constexpr (STAGES > 1)
      __pipeline_commit();
  }
  if constexpr (STAGES > 1)
    __pipeline_wait_prior(STAGES - 2);
  __syncthreads();

  share_to_reg_one_stage_A<CTA_M, CTA_N, CTA_K, STAGES, WARP_M / INTRIN_M>(A_shared, A_shared_warp_[0], warp_offset_m, warp_offset_n, warp_offset_k, 0);
  share_to_reg_one_stage_B<CTA_M, CTA_N, CTA_K, STAGES, true, WARP_N / INTRIN_N, G>(B_shared, scales_shared, zeros_shared, B_shared_warp_tmp_[0], B_shared_warp_[0], warp_offset_m, warp_offset_n, warp_offset_k, 0);
  constexpr int SHARED_K_ITERS = WARP_K / INTRIN_K;

  for (; k_0_0 < gemm_iters; ++k_0_0, ++k_0_0_ld)
  {
    int ld_stage = k_0_0_ld % STAGES;
    int compute_stage = k_0_0 % STAGES;
    half *A_shared_this_compute_stage;
    half *B_shared_this_compute_stage;
    half *scales_shared_this_compute_stage;
    half *zeros_shared_this_compute_stage;

#pragma unroll
    for (int iter_k = 0; iter_k < SHARED_K_ITERS; ++iter_k)
    {
      A_shared_this_compute_stage = A_shared + compute_stage * kSmemSizeAPerStage;
      B_shared_this_compute_stage = B_shared + compute_stage * kSmemSizeBPerStage;
      scales_shared_this_compute_stage = scales_shared + (compute_stage / scales_load_interval * scales_per_load) * CTA_N;
      zeros_shared_this_compute_stage = zeros_shared + (compute_stage / scales_load_interval * scales_per_load) * CTA_N;
      share_to_reg_one_stage_A<CTA_M, CTA_N, CTA_K, STAGES, WARP_M / INTRIN_M>(A_shared_this_compute_stage, A_shared_warp_[(iter_k + 1) % 2], warp_offset_m, warp_offset_n, warp_offset_k, (iter_k + 1) % SHARED_K_ITERS);
      if ((iter_k + 1) % kInterleave == 0)
      {
        if (compute_stage % 2 == 1)
        {
          share_to_reg_one_stage_B<CTA_M, CTA_N, CTA_K, STAGES, true, WARP_N / INTRIN_N, G>(
              B_shared_this_compute_stage, scales_shared_this_compute_stage, zeros_shared_this_compute_stage,
              B_shared_warp_tmp_[1], B_shared_warp_[((iter_k + 1) / 2) % 2],
              warp_offset_m, warp_offset_n, warp_offset_k, (iter_k + 1) % SHARED_K_ITERS);
        }
        else
        {
          share_to_reg_one_stage_B<CTA_M, CTA_N, CTA_K, STAGES, true, WARP_N / INTRIN_N, G>(
              B_shared_this_compute_stage, scales_shared_this_compute_stage, zeros_shared_this_compute_stage,
              B_shared_warp_tmp_[0], B_shared_warp_[((iter_k + 1) / 2) % 2],
              warp_offset_m, warp_offset_n, warp_offset_k, (iter_k + 1) % SHARED_K_ITERS);
        }
      }
      else
      {
        if (compute_stage % 2 == 1)
        {
          share_to_reg_one_stage_B<CTA_M, CTA_N, CTA_K, STAGES, false, WARP_N / INTRIN_N, G>(
              B_shared_this_compute_stage, scales_shared_this_compute_stage, zeros_shared_this_compute_stage,
              B_shared_warp_tmp_[1], B_shared_warp_[((iter_k + 1) / 2) % 2],
              warp_offset_m, warp_offset_n, warp_offset_k, (iter_k + 1) % SHARED_K_ITERS);
        }
        else
        {
          share_to_reg_one_stage_B<CTA_M, CTA_N, CTA_K, STAGES, false, WARP_N / INTRIN_N, G>(
              B_shared_this_compute_stage, scales_shared_this_compute_stage, zeros_shared_this_compute_stage,
              B_shared_warp_tmp_[0], B_shared_warp_[((iter_k + 1) / 2) % 2],
              warp_offset_m, warp_offset_n, warp_offset_k, (iter_k + 1) % SHARED_K_ITERS);
        }
      }
      half *A_shared_warp = A_shared_warp_[iter_k % 2];
      half *B_shared_warp = B_shared_warp_[(iter_k / 2) % 2];

      for (int i_0_3 = 0; i_0_3 < WARP_M / INTRIN_M; ++i_0_3)
      {
        for (int j_0_4 = 0; j_0_4 < WARP_N / INTRIN_N; ++j_0_4)
        {
          mma_m16n8k16(C_warp + i_0_3 * WARP_N / INTRIN_N * 8 + j_0_4 * 8, A_shared_warp + i_0_3 * 8, B_shared_warp + j_0_4 * 16 + (iter_k % 2) * 4);
          mma_m16n8k16(C_warp + i_0_3 * WARP_N / INTRIN_N * 8 + j_0_4 * 8 + 4, A_shared_warp + i_0_3 * 8, B_shared_warp + j_0_4 * 16 + (iter_k % 2) * 4 + 8);
        }
      }

      if (iter_k < WARP_K / INTRIN_K - 1)
      {
        if constexpr (STAGES == 1)
          __syncthreads();
        global_to_share_one_stage_A<CTA_M, CTA_N, CTA_K, CTA_SIZE, WARP_K / INTRIN_K, STAGES>(A, A_shared + ld_stage * kSmemSizeAPerStage, M, K, cta_offset_m, cta_offset_n, cta_offset_k, k_0_0_ld, iter_k, k_0_0_ld < gemm_iters);
        global_to_share_one_stage_B<CTA_M, CTA_N, CTA_K, CTA_SIZE, WARP_K / INTRIN_K, STAGES>(B, B_shared + ld_stage * kSmemSizeBPerStage, K, cta_offset_m, cta_offset_n, cta_offset_k, k_0_0_ld, iter_k, k_0_0_ld < gemm_iters);
      }

      if (iter_k == WARP_K / INTRIN_K - 2)
      {
        if constexpr (STAGES == 1 && WARP_K / INTRIN_K > 2)
        {
          __syncthreads();
        }
        global_to_share_one_stage_A<CTA_M, CTA_N, CTA_K, CTA_SIZE, WARP_K / INTRIN_K, STAGES>(A, A_shared + ld_stage * kSmemSizeAPerStage, M, K, cta_offset_m, cta_offset_n, cta_offset_k, k_0_0_ld, iter_k + 1, k_0_0_ld < gemm_iters);
        global_to_share_one_stage_B<CTA_M, CTA_N, CTA_K, CTA_SIZE, WARP_K / INTRIN_K, STAGES>(B, B_shared + ld_stage * kSmemSizeBPerStage, K, cta_offset_m, cta_offset_n, cta_offset_k, k_0_0_ld, iter_k + 1, k_0_0_ld < gemm_iters);
        global_to_share_one_stage_scales<CTA_M, CTA_N, CTA_K, CTA_SIZE, STAGES, G>(
            scales, scales_shared + (ld_stage / scales_load_interval * scales_per_load) * CTA_N,
            zeros, zeros_shared + (ld_stage / scales_load_interval * scales_per_load) * CTA_N,
            N, cta_offset_m, cta_offset_n, cta_offset_k,
            k_0_0_ld, iter_k, k_0_0_ld < gemm_iters && k_0_0_ld % scales_load_interval == 0);
        if constexpr (STAGES > 1)
        {
          __pipeline_commit();
          __pipeline_wait_prior(STAGES - 2);
        }
        compute_stage = (k_0_0 + 1) % STAGES;
        __syncthreads();
      }
    }
  }
  __pipeline_commit();
  __pipeline_wait_prior(0);
  __syncthreads();
  if constexpr (SLICES > 1)
  {
#pragma unroll
    for (int z = 0; z < SLICES; ++z)
    {
      if (slice_id == z)
      {
#pragma unroll
        for (int ax0_0_1 = 0; ax0_0_1 < WARP_M / INTRIN_M; ++ax0_0_1)
        {
#pragma unroll
          for (int ax1_0_1 = 0; ax1_0_1 < WARP_N / INTRIN_N; ++ax1_0_1)
          {
#pragma unroll
            for (int local_id = 0; local_id < OP_M * 16 / WARP_SIZE; ++local_id)
            {
              if (z > 0)
              {
                C_warp[ax0_0_1 * WARP_N / INTRIN_N * 8 + ax1_0_1 * 8 + local_id] += C_shared[warp_offset_m * CTA_N + ax0_0_1 * OP_M * CTA_N + warp_offset_n + ax1_0_1 * 16 + ((local_id % 4) / 2 * 8 + (threadIdx.x / 4)) * CTA_N + (local_id / 4) * 8 + (local_id % 2) + (threadIdx.x % 4) * 2];
              }
              C_shared[warp_offset_m * CTA_N + ax0_0_1 * OP_M * CTA_N + warp_offset_n + ax1_0_1 * 16 + ((local_id % 4) / 2 * 8 + (threadIdx.x / 4)) * CTA_N + (local_id / 4) * 8 + (local_id % 2) + (threadIdx.x % 4) * 2] = C_warp[ax0_0_1 * WARP_N / INTRIN_N * 8 + ax1_0_1 * 8 + local_id];
            };
          }
        }
      }
      __syncthreads();
    }
    if (slice_id == 0)
    {
#pragma unroll
      for (int ax0_0_1 = 0; ax0_0_1 < WARP_M / INTRIN_M; ++ax0_0_1)
      {
#pragma unroll
        for (int ax1_0_1 = 0; ax1_0_1 < WARP_N / INTRIN_N; ++ax1_0_1)
        {
#pragma unroll
          for (int local_id = 0; local_id < OP_M * 16 / WARP_SIZE; ++local_id)
          {
            C_warp[ax0_0_1 * WARP_N / INTRIN_N * 8 + ax1_0_1 * 8 + local_id] = C_shared[warp_offset_m * CTA_N + ax0_0_1 * OP_M * CTA_N + warp_offset_n + ax1_0_1 * 16 + ((local_id % 4) / 2 * 8 + (threadIdx.x / 4)) * CTA_N + (local_id / 4) * 8 + (local_id % 2) + (threadIdx.x % 4) * 2];
          };
        }
      }
    }
  }

  if (slice_id == 0)
  {
    Semaphore semaphore(semaphores + blockIdx_y, threadIdx.x);

    if constexpr (SPLITK > 1)
    {
      semaphore.fetch();
    }

    if (blockIdx_z != 0)
    {
      semaphore.wait(blockIdx_z);
      for (int ax0_0_1 = 0; ax0_0_1 < WARP_M / INTRIN_M; ++ax0_0_1)
      {
        for (int ax1_0_1 = 0; ax1_0_1 < WARP_N / INTRIN_N; ++ax1_0_1)
        {
          for (int local_id = 0; local_id < OP_M * 16 / WARP_SIZE; local_id += 2)
          {
            int write_row = cta_offset_m + warp_offset_m + ax0_0_1 * OP_M + ((local_id % 4) / 2 * 8 + (threadIdx.x / 4));

            if (write_row < M)
            {
              half2 *existing_psum_ptr = reinterpret_cast<half2 *>(
                  C + write_row * N +
                  cta_offset_n + warp_offset_n + ax1_0_1 * 16 +
                  (local_id / 4) * 8 + (local_id % 2) + (threadIdx.x % 4) * 2);

              *existing_psum_ptr = __hadd2(*existing_psum_ptr,
                                           __float22half2_rn(*reinterpret_cast<float2 *>(C_warp + ax0_0_1 * WARP_N / INTRIN_N * 8 +
                                                                                         ax1_0_1 * 8 + local_id)));
            }
          };
        }
      }
    }
    else
    {
      for (int ax0_0_1 = 0; ax0_0_1 < WARP_M / INTRIN_M; ++ax0_0_1)
      {
        for (int ax1_0_1 = 0; ax1_0_1 < WARP_N / INTRIN_N; ++ax1_0_1)
        {
          for (int local_id = 0; local_id < OP_M * 16 / WARP_SIZE; local_id += 2)
          {
            int write_row = cta_offset_m + warp_offset_m + ax0_0_1 * OP_M + ((local_id % 4) / 2 * 8 + (threadIdx.x / 4));
            if (write_row < M)
            {
              *reinterpret_cast<half2 *>(
                  C + write_row * N +
                  cta_offset_n + warp_offset_n + ax1_0_1 * 16 +
                  (local_id / 4) * 8 + (local_id % 2) + (threadIdx.x % 4) * 2) =
                  __float22half2_rn(*reinterpret_cast<float2 *>(C_warp + ax0_0_1 * WARP_N / INTRIN_N * 8 +
                                                                ax1_0_1 * 8 + local_id));
            }
          };
        }
      }
    }

    if constexpr (SPLITK > 1)
    {

      int lock = 0;
      if (SPLITK == blockIdx_z + 1)
      {

        lock = 0;
      }
      else
      {
        lock = blockIdx_z + 1;
      }
      semaphore.release(lock);
    }
  }
}

template <int CTA_M, int CTA_N, int CTA_K, int CTA_SIZE, int SHARED_K_ITERS, int STAGES>
__device__ __inline__ void global_to_share_one_stage_A_T2(half *src, half *dst, int global_nrows, int global_ncols, int cta_offset_m, int cta_offset_n, int global_iter_k, int shared_iter_k, bool mask)
{
  constexpr int threads_needed = (CTA_M * CTA_K) / PACK_SIZE / SHARED_K_ITERS;
  constexpr int threads_used = threads_needed < CTA_SIZE ? threads_needed : CTA_SIZE;
  constexpr int total_global_iters = (CTA_M * CTA_K) / PACK_SIZE / threads_used;
  constexpr int partial_global_iters = (total_global_iters + SHARED_K_ITERS - 1) / SHARED_K_ITERS;
  constexpr int cta_step_m_or_n = (threads_used * PACK_SIZE) / CTA_K;
  constexpr int warp_step_m_or_n = (WARP_SIZE * PACK_SIZE) / CTA_K;
  constexpr int threads_per_row = CTA_K / PACK_SIZE;
  constexpr int kSmemCol = CTA_K + SMEM_PAD_A;
  bool local_mask = mask & (threadIdx.y * WARP_SIZE + threadIdx.x < threads_used);
  int ld_col = (threadIdx.x % threads_per_row);
#pragma unroll
  for (int _global_iter = 0; _global_iter < partial_global_iters; ++_global_iter)
  {
    int global_iter = shared_iter_k * partial_global_iters + _global_iter;
    int ld_row = global_iter * cta_step_m_or_n + threadIdx.y * warp_step_m_or_n + (threadIdx.x / threads_per_row);
    int ld_col_swizzled = (ld_col ^ (ld_row) & 7) * PACK_SIZE;
    void *dst_ptr = (void *)(dst + ld_row * kSmemCol + ld_col_swizzled);
    uint4 *src_ptr = (uint4 *)(src + (ld_row + cta_offset_m) * global_ncols + ld_col * PACK_SIZE + global_iter_k * CTA_K); // cta_offset_m * global_ncols + global_iter * cta_step_m_or_n * global_ncols + threadIdx.y * warp_step_m_or_n * global_ncols + (threadIdx.x / threads_per_row) * global_ncols + global_iter_k * CTA_K + (threadIdx.x % threads_per_row) * PACK_SIZE);
    if constexpr (STAGES > 1)
    {
      uint32_t addr = cast_smem_ptr_to_uint(dst_ptr);
      cp_async_cg_A(addr, src_ptr, local_mask & (ld_row + cta_offset_m < global_nrows));
    }
    else
    {
      if (local_mask & (ld_row + cta_offset_m < global_nrows))
        *(uint4 *)dst_ptr = *src_ptr;
    }
  }
}

template <int CTA_M, int CTA_N, int CTA_K, int CTA_SIZE, int SHARED_K_ITERS, int STAGES>
__device__ __inline__ void global_to_share_one_stage_B_T2(half *src, half *dst, int global_ncols, int cta_offset_m, int cta_offset_n, int global_iter_k, int shared_iter_k, bool mask)
{
  constexpr int threads_needed = (CTA_N / kInterleave * CTA_K) / PACK_SIZE / SHARED_K_ITERS;
  constexpr int threads_used = threads_needed < CTA_SIZE ? threads_needed : CTA_SIZE;
  constexpr int total_global_iters = (CTA_N / kInterleave * CTA_K) / PACK_SIZE / threads_used;
  constexpr int partial_global_iters = (total_global_iters + SHARED_K_ITERS - 1) / SHARED_K_ITERS;
  constexpr int cta_step_m_or_n = (threads_used * PACK_SIZE) / CTA_K;
  constexpr int warp_step_m_or_n = (WARP_SIZE * PACK_SIZE) / CTA_K;
  constexpr int threads_per_row = CTA_K / PACK_SIZE;
  constexpr int kSmemCol = CTA_K + SMEM_PAD_B;
  bool local_mask = mask & (threadIdx.y * WARP_SIZE + threadIdx.x < threads_used);
#pragma unroll
  for (int _global_iter = 0; _global_iter < partial_global_iters; ++_global_iter)
  {
    int global_iter = shared_iter_k * partial_global_iters + _global_iter;

    int ld_row = global_iter * cta_step_m_or_n + threadIdx.y * warp_step_m_or_n + (threadIdx.x / threads_per_row);
    int ld_col = (threadIdx.x % threads_per_row);
    int ld_col_swizzled = ld_col ^ (ld_row % 2) & 7;
    void *dst_ptr = (void *)(dst + (ld_row * kSmemCol + ld_col_swizzled * PACK_SIZE));
    uint4 *src_ptr = (uint4 *)(src + global_iter_k * CTA_K + cta_offset_n / kInterleave * global_ncols + ld_row * global_ncols + ld_col * PACK_SIZE);
    if constexpr (STAGES > 1)
    {
      uint32_t addr = cast_smem_ptr_to_uint(dst_ptr);
      cp_async_cg_A(addr, src_ptr, local_mask);
    }
    else
    {
      if (local_mask)
        *(uint4 *)dst_ptr = *src_ptr;
    }
  }
}

template <int CTA_M, int CTA_N, int CTA_K, int CTA_SIZE, int STAGES, int G>
__device__ __inline__ void global_to_share_one_stage_scales_T2(half *src, half *dst, half *src_z, half *dst_z, int global_ncols, int cta_offset_m, int cta_offset_n, int global_iter_k, int shared_iter_k, bool mask)
{
  constexpr int threads_needed = CTA_N / PACK_SIZE / 1;
  constexpr int threads_used = threads_needed < CTA_SIZE ? threads_needed : CTA_SIZE;
  constexpr int total_global_iters = CTA_N / PACK_SIZE / threads_used;
  constexpr int threads_per_row = CTA_N / PACK_SIZE;
  constexpr int kSmemCol = CTA_N;
  bool local_mask = mask & (threadIdx.y * WARP_SIZE + threadIdx.x < threads_used);
  int g_idx = global_iter_k * CTA_K / G;

  void *dst_ptr = (void *)(dst + (threadIdx.x % threads_per_row) * PACK_SIZE);
  uint4 *src_ptr = (uint4 *)(src + g_idx * global_ncols + cta_offset_n + (threadIdx.x % threads_per_row) * PACK_SIZE);
  void *dst_ptr_z = (void *)(dst_z + (threadIdx.x % threads_per_row) * PACK_SIZE);
  uint4 *src_ptr_z = (uint4 *)(src_z + g_idx * global_ncols + cta_offset_n + (threadIdx.x % threads_per_row) * PACK_SIZE);
  if (STAGES > 1)
  {
    uint32_t addr = cast_smem_ptr_to_uint(dst_ptr);
    cp_async_cg_A(addr, src_ptr, local_mask);
    uint32_t addr_z = cast_smem_ptr_to_uint(dst_ptr_z);
    cp_async_cg_A(addr_z, src_ptr_z, local_mask);
  }
  else
  {
    if (local_mask)
    {
      *(uint4 *)dst_ptr = *src_ptr;
      *(uint4 *)dst_ptr_z = *src_ptr_z;
    }
  }
}

template <int CTA_M, int CTA_N, int CTA_K, int STAGES, int shared_iters>
__device__ __inline__ void share_to_reg_one_stage_A_T2(half *src, half *dst, int warp_offset_m, int warp_offset_n, int k_0_1)
{
  constexpr int kSmemCol = CTA_K + SMEM_PAD_A;

  for (int shared_iter = 0; shared_iter < shared_iters; ++shared_iter)
  {

    int ld_row = warp_offset_m + shared_iter * OP_M + (threadIdx.x % 16);
    int ld_col = k_0_1 * 16 + (threadIdx.x / 16) * 8;
    int ld_col_swizzled = ((ld_col / PACK_SIZE) ^ (ld_row) & 7) * PACK_SIZE;
    void *addr_ptr = (void *)(src + ld_row * kSmemCol + ld_col_swizzled);

    uint32_t addr = cast_smem_ptr_to_uint(addr_ptr);
    ldmatrix_m8n8_x4_b16(dst, shared_iter, addr);
  }
}

template <int CTA_M, int CTA_N, int CTA_K, int STAGES, bool ldmatrix, int shared_iters, int G>
__device__ __inline__ void share_to_reg_one_stage_B_T2(half *src, half *src_scales, half *src_zeros, half *dst, half *dst_fp16, int warp_offset_m, int warp_offset_n, int k_0_1)
{
  constexpr int kSmemCol = CTA_K + SMEM_PAD_B;
  int r0 = ((threadIdx.x / 8 / 2) * 8 + threadIdx.x % 8);
  int c0 = ((threadIdx.x / 8) % 2) * 8;
  int r = r0 / 4;
  int c = (r0 % 4) * 16 + c0;
  int c_swizzled = ((c / PACK_SIZE) ^ (r % 2) & 7) * PACK_SIZE;

  if constexpr (ldmatrix)
  {
#pragma unroll
    for (int shared_iter = 0; shared_iter < shared_iters; ++shared_iter)
    {
      void *addr_ptr = (void *)(src + warp_offset_n / kInterleave * kSmemCol + shared_iter * 16 / kInterleave * kSmemCol + k_0_1 * 16 + r * kSmemCol + c_swizzled);
      uint32_t addr = cast_smem_ptr_to_uint(addr_ptr);
      ldmatrix_m8n8_x4_b16(dst, shared_iter, addr);
    }
  }

#pragma unroll
  for (int shared_iter = 0; shared_iter < shared_iters; ++shared_iter)
  {
    half scale = src_scales[warp_offset_n + 16 * shared_iter + 8 * (k_0_1 % 2) + threadIdx.x / 4];
    half zero = src_zeros[warp_offset_n + 16 * shared_iter + 8 * (k_0_1 % 2) + threadIdx.x / 4];
    half2 scale2 = make_half2(scale, scale);
    half2 zero2 = make_half2(zero, zero);
    half2 loaded[4];
    dequantize_s4_to_fp16x2(*reinterpret_cast<half2 *>(dst + (k_0_1 % 2) * 4 + (k_0_1 / 2 * 2) + shared_iter * 8), reinterpret_cast<uint4 *>(loaded));
#pragma unroll
    for (int i = 0; i < 4; i++)
    {
      loaded[i] = __hfma2(loaded[i], scale2, zero2);
    }
    *reinterpret_cast<uint4 *>(dst_fp16 + shared_iter * 16 + 8 * (k_0_1 % 2)) = *reinterpret_cast<uint4 *>(loaded);
  }
}

template <int CTA_M, int CTA_N, int CTA_K, int WARP_M, int WARP_N, int WARP_K, int STAGES, int G>
__global__ void gemm_w4a16_T2(half *__restrict__ A, half *__restrict__ B, half *__restrict__ scales, half *__restrict__ zeros, half *__restrict__ C, int M, int N, int K)
{
  constexpr int NUM_WARPS = CTA_M / WARP_M * CTA_N / WARP_N;
  constexpr int CTA_SIZE = NUM_WARPS * WARP_SIZE;
  int num_blocks_n = (N + CTA_N - 1) / CTA_N;
  int num_blocks_m = (M + CTA_M - 1) / CTA_M;
  int blockIdx_x = 0;
  int blockIdx_y = blockIdx.x % (num_blocks_m * num_blocks_n);
  int blockIdx_z = blockIdx.x / (num_blocks_m * num_blocks_n);
  const int log_tile = get_log_tile<1>((N + CTA_N - 1) / CTA_N);
  int blockIdx_m = blockIdx_y / (num_blocks_n >> log_tile);
  int blockIdx_n = blockIdx_y % (num_blocks_n >> log_tile);
  const uint2 block_idx_mapping = get_block_idx_mapping(blockIdx_m, blockIdx_n, log_tile);
  blockIdx_m = block_idx_mapping.x;
  blockIdx_n = block_idx_mapping.y;

  float C_warp[CTA_M * CTA_N / CTA_SIZE];
  constexpr int kSmemPadKA = CTA_K + SMEM_PAD_A;
  constexpr int kSmemPadKB = CTA_K + SMEM_PAD_B;
  constexpr int kSmemSizeAPerStage = CTA_M * kSmemPadKA;
  constexpr int kSmemSizeBPerStage = CTA_N / kInterleave * kSmemPadKB;
  constexpr int kSmemSizeA = kSmemSizeAPerStage * STAGES;
  constexpr int kSmemSizeB = kSmemSizeBPerStage * STAGES;
  constexpr int kSmemSizeScales = CTA_N * STAGES / 2;
  constexpr int kSmemSizeZeros = CTA_N * STAGES / 2;
  constexpr int scales_load_interval = G / CTA_K;
  extern __shared__ half mem_shared[];
  half *A_shared = mem_shared;
  half *B_shared = mem_shared + kSmemSizeA;
  half *scales_shared = mem_shared + kSmemSizeA + kSmemSizeB;
  half *zeros_shared = mem_shared + kSmemSizeA + kSmemSizeB + kSmemSizeScales;
  half A_shared_warp_[2][WARP_M * INTRIN_K /
                         WARP_SIZE];
  half B_shared_warp_[2][WARP_N * 32 /
                         WARP_SIZE];
  half B_shared_warp_tmp_[2][WARP_N * 16 /
                             WARP_SIZE];
  int cta_offset_m = blockIdx_m * CTA_M;
  int cta_offset_n = blockIdx_n * CTA_N;
  int warp_offset_m = (threadIdx.y % (CTA_M / WARP_M)) * WARP_M;
  int warp_offset_n = (threadIdx.y / (CTA_M / WARP_M)) * WARP_N;

  for (int i = 0; i < CTA_M * CTA_N / CTA_SIZE; i++)
    C_warp[i] = 0.0;

  int gemm_iters = (K + CTA_K - 1) / CTA_K;
  int k_0_0_ld = 0;
  int k_0_0 = 0;
  constexpr int prologue_stages = STAGES == 1 ? 1 : STAGES - 1;
#pragma unroll
  for (k_0_0_ld = 0; k_0_0_ld < prologue_stages; ++k_0_0_ld)
  {
    global_to_share_one_stage_A_T2<CTA_M, CTA_N, CTA_K, CTA_SIZE, 1, STAGES>(A, A_shared + k_0_0_ld * kSmemSizeAPerStage, M, K, cta_offset_m, cta_offset_n, k_0_0_ld, 0, true);
    global_to_share_one_stage_B_T2<CTA_M, CTA_N, CTA_K, CTA_SIZE, 1, STAGES>(B, B_shared + k_0_0_ld * kSmemSizeBPerStage, K, cta_offset_m, cta_offset_n, k_0_0_ld, 0, true);
    global_to_share_one_stage_scales_T2<CTA_M, CTA_N, CTA_K, CTA_SIZE, STAGES, G>(
        scales, scales_shared + (k_0_0_ld / scales_load_interval) * CTA_N,
        zeros, zeros_shared + (k_0_0_ld / scales_load_interval) * CTA_N,
        N, cta_offset_m, cta_offset_n, k_0_0_ld, 0, k_0_0_ld < gemm_iters && k_0_0_ld % scales_load_interval == 0);
    if constexpr (STAGES > 1)
      __pipeline_commit();
  }
  if constexpr (STAGES > 1)
    __pipeline_wait_prior(STAGES - 2);
  __syncthreads();

  share_to_reg_one_stage_A_T2<CTA_M, CTA_N, CTA_K, STAGES, WARP_M / INTRIN_M>(A_shared, A_shared_warp_[0], warp_offset_m, warp_offset_n, 0);
  share_to_reg_one_stage_B_T2<CTA_M, CTA_N, CTA_K, STAGES, true, WARP_N / INTRIN_N, G>(B_shared, scales_shared, zeros_shared, B_shared_warp_tmp_[0], B_shared_warp_[0], warp_offset_m, warp_offset_n, 0);
  constexpr int SHARED_K_ITERS = WARP_K / INTRIN_K;

  for (; k_0_0 < gemm_iters; ++k_0_0, ++k_0_0_ld)
  {
    int ld_stage = k_0_0_ld % STAGES;
    int compute_stage = k_0_0 % STAGES;
    half *A_shared_this_compute_stage;
    half *B_shared_this_compute_stage;
    half *scales_shared_this_compute_stage;
    half *zeros_shared_this_compute_stage;

    for (int iter_k = 0; iter_k < SHARED_K_ITERS; ++iter_k)
    {
      A_shared_this_compute_stage = A_shared + compute_stage * kSmemSizeAPerStage;
      B_shared_this_compute_stage = B_shared + compute_stage * kSmemSizeBPerStage;
      scales_shared_this_compute_stage = scales_shared + (compute_stage / scales_load_interval) * CTA_N;
      zeros_shared_this_compute_stage = zeros_shared + (compute_stage / scales_load_interval) * CTA_N;
      share_to_reg_one_stage_A_T2<CTA_M, CTA_N, CTA_K, STAGES, WARP_M / INTRIN_M>(A_shared_this_compute_stage, A_shared_warp_[(iter_k + 1) % 2], warp_offset_m, warp_offset_n, (iter_k + 1) % SHARED_K_ITERS);
      if ((iter_k + 1) % kInterleave == 0)
      {
        if (compute_stage % 2 == 1)
        {
          share_to_reg_one_stage_B_T2<CTA_M, CTA_N, CTA_K, STAGES, true, WARP_N / INTRIN_N, G>(
              B_shared_this_compute_stage, scales_shared_this_compute_stage, zeros_shared_this_compute_stage,
              B_shared_warp_tmp_[1], B_shared_warp_[((iter_k + 1) / 2) % 2],
              warp_offset_m, warp_offset_n, (iter_k + 1) % SHARED_K_ITERS);
        }
        else
        {
          share_to_reg_one_stage_B_T2<CTA_M, CTA_N, CTA_K, STAGES, true, WARP_N / INTRIN_N, G>(
              B_shared_this_compute_stage, scales_shared_this_compute_stage, zeros_shared_this_compute_stage,
              B_shared_warp_tmp_[0], B_shared_warp_[((iter_k + 1) / 2) % 2],
              warp_offset_m, warp_offset_n, (iter_k + 1) % SHARED_K_ITERS);
        }
      }
      else
      {
        if (compute_stage % 2 == 1)
        {
          share_to_reg_one_stage_B_T2<CTA_M, CTA_N, CTA_K, STAGES, false, WARP_N / INTRIN_N, G>(
              B_shared_this_compute_stage, scales_shared_this_compute_stage, zeros_shared_this_compute_stage,
              B_shared_warp_tmp_[1], B_shared_warp_[((iter_k + 1) / 2) % 2],
              warp_offset_m, warp_offset_n, (iter_k + 1) % SHARED_K_ITERS);
        }
        else
        {
          share_to_reg_one_stage_B_T2<CTA_M, CTA_N, CTA_K, STAGES, false, WARP_N / INTRIN_N, G>(
              B_shared_this_compute_stage, scales_shared_this_compute_stage, zeros_shared_this_compute_stage,
              B_shared_warp_tmp_[0], B_shared_warp_[((iter_k + 1) / 2) % 2],
              warp_offset_m, warp_offset_n, (iter_k + 1) % SHARED_K_ITERS);
        }
      }
      __syncthreads();
      half *A_shared_warp = A_shared_warp_[iter_k % 2];
      half *B_shared_warp = B_shared_warp_[(iter_k / 2) % 2];
      for (int i_0_3 = 0; i_0_3 < WARP_M / INTRIN_M; ++i_0_3)
      {
        for (int j_0_4 = 0; j_0_4 < WARP_N / INTRIN_N; ++j_0_4)
        {
          mma_m16n8k16(C_warp + i_0_3 * WARP_N / INTRIN_N * 8 + j_0_4 * 8, A_shared_warp + i_0_3 * 8, B_shared_warp + j_0_4 * 16 + (iter_k % 2) * 4);
          mma_m16n8k16(C_warp + i_0_3 * WARP_N / INTRIN_N * 8 + j_0_4 * 8 + 4, A_shared_warp + i_0_3 * 8, B_shared_warp + j_0_4 * 16 + (iter_k % 2) * 4 + 8);
        }
      }

      if (iter_k < WARP_K / INTRIN_K - 1)
      {
        if constexpr (STAGES == 1)
          __syncthreads();
        global_to_share_one_stage_A_T2<CTA_M, CTA_N, CTA_K, CTA_SIZE, WARP_K / INTRIN_K, STAGES>(A, A_shared + ld_stage * kSmemSizeAPerStage, M, K, cta_offset_m, cta_offset_n, k_0_0_ld, iter_k, k_0_0_ld < gemm_iters);
        global_to_share_one_stage_B_T2<CTA_M, CTA_N, CTA_K, CTA_SIZE, WARP_K / INTRIN_K, STAGES>(B, B_shared + ld_stage * kSmemSizeBPerStage, K, cta_offset_m, cta_offset_n, k_0_0_ld, iter_k, k_0_0_ld < gemm_iters);
      }

      if (iter_k == WARP_K / INTRIN_K - 2)
      {
        if constexpr (STAGES == 1 && WARP_K / INTRIN_K > 2)
        {
          __syncthreads();
        }
        global_to_share_one_stage_A_T2<CTA_M, CTA_N, CTA_K, CTA_SIZE, WARP_K / INTRIN_K, STAGES>(A, A_shared + ld_stage * kSmemSizeAPerStage, M, K, cta_offset_m, cta_offset_n, k_0_0_ld, iter_k + 1, k_0_0_ld < gemm_iters);
        global_to_share_one_stage_B_T2<CTA_M, CTA_N, CTA_K, CTA_SIZE, WARP_K / INTRIN_K, STAGES>(B, B_shared + ld_stage * kSmemSizeBPerStage, K, cta_offset_m, cta_offset_n, k_0_0_ld, iter_k + 1, k_0_0_ld < gemm_iters);
        global_to_share_one_stage_scales_T2<CTA_M, CTA_N, CTA_K, CTA_SIZE, STAGES, G>(
            scales, scales_shared + (ld_stage / scales_load_interval) * CTA_N,
            zeros, zeros_shared + (ld_stage / scales_load_interval) * CTA_N,
            N, cta_offset_m, cta_offset_n, k_0_0_ld, iter_k, k_0_0_ld < gemm_iters && k_0_0_ld % scales_load_interval == 0);
        if constexpr (STAGES > 1)
        {
          __pipeline_commit();
          __pipeline_wait_prior(STAGES - 2);
        }
        compute_stage = (k_0_0 + 1) % STAGES;
        __syncthreads();
      }
    }
  }
  for (int ax0_0_1 = 0; ax0_0_1 < WARP_M / INTRIN_M; ++ax0_0_1)
  {
    for (int ax1_0_1 = 0; ax1_0_1 < WARP_N / INTRIN_N; ++ax1_0_1)
    {
      for (int local_id = 0; local_id < OP_M * 16 / WARP_SIZE; local_id += 2)
      {
        int write_row = cta_offset_m + warp_offset_m + ax0_0_1 * OP_M + ((local_id % 4) / 2 * 8 + (threadIdx.x / 4));
        if (write_row < M)
        {
          *reinterpret_cast<half2 *>(
              C + write_row * N +
              cta_offset_n + warp_offset_n + ax1_0_1 * 16 +
              (local_id / 4) * 8 + (local_id % 2) + (threadIdx.x % 4) * 2) =
              __float22half2_rn(*reinterpret_cast<float2 *>(C_warp + ax0_0_1 * WARP_N / INTRIN_N * 8 +
                                                            ax1_0_1 * 8 + local_id));
        }
      };
    }
  }
}

torch::Tensor gemm_forward_cuda_new(
    torch::Tensor _in_feats,
    torch::Tensor _kernel,
    torch::Tensor _scales,
    torch::Tensor _zeros)
{
  std::vector<int64_t> output_shape = _in_feats.sizes().vec();
  output_shape.back() = _kernel.size(0) * kInterleave;
  int num_in_feats = _in_feats.numel() / _in_feats.size(-1);
  int num_in_channels = _in_feats.size(-1);
  auto in_feats = reinterpret_cast<half *>(_in_feats.data_ptr<at::Half>());
  auto kernel = reinterpret_cast<half *>(_kernel.data_ptr<int16_t>());
  auto scales = reinterpret_cast<half *>(_scales.data_ptr<at::Half>());
  auto zeros = reinterpret_cast<half *>(_zeros.data_ptr<at::Half>());
  auto options =
      torch::TensorOptions().dtype(_in_feats.dtype()).device(_in_feats.device());
  auto options_int =
      torch::TensorOptions().dtype(torch::kInt32).device(_in_feats.device());
  at::Tensor _out_feats = torch::empty(output_shape, options);
  int num_out_feats = _out_feats.numel() / _out_feats.size(-1);
  int num_out_channels = _out_feats.size(-1);
  auto out_feats = reinterpret_cast<half *>(_out_feats.data_ptr<at::Half>());

  if (num_out_feats <= 32)
  {
    constexpr int G = 128;
    constexpr int CTA_M = 16;
    constexpr int CTA_N = 128;
    constexpr int CTA_K = 128;
    constexpr int WARP_M = 16;
    constexpr int WARP_N = 32;
    constexpr int WARP_K = 64;
    constexpr int SPLITK = 2;
    constexpr int STAGES = 4;
    KERNEL_LAUNCH_CODE
  }
  else if (num_out_feats <= 64)
  {

    constexpr int G = 128;
    constexpr int CTA_M = 16;
    constexpr int CTA_N = 128;
    constexpr int CTA_K = 128;
    constexpr int WARP_M = 16;
    constexpr int WARP_N = 32;
    constexpr int WARP_K = 64;
    constexpr int SPLITK = 1;
    constexpr int STAGES = 3;
    KERNEL_LAUNCH_CODE
  }
  else if (num_out_feats <= 128)
  {
    constexpr int G = 128;
    constexpr int CTA_M = 32;
    constexpr int CTA_N = 128;
    constexpr int CTA_K = 128;
    constexpr int WARP_M = 32;
    constexpr int WARP_N = 32;
    constexpr int WARP_K = 64;
    constexpr int SPLITK = 1;
    constexpr int STAGES = 4;
    KERNEL_LAUNCH_CODE
  }
  else if (num_out_feats <= 192)
  {
    constexpr int G = 128;
    constexpr int CTA_M = 64;
    constexpr int CTA_N = 128;
    constexpr int CTA_K = 64;
    constexpr int WARP_M = 64;
    constexpr int WARP_N = 32;
    constexpr int WARP_K = 64;
    constexpr int SPLITK = 1;
    constexpr int STAGES = 4;
    KERNEL_LAUNCH_CODE
  }
  else
  {
    constexpr int G = 128;
    constexpr int CTA_M = 64;
    constexpr int CTA_N = 128;
    constexpr int CTA_K = 64;
    constexpr int WARP_M = 64;
    constexpr int WARP_N = 32;
    constexpr int WARP_K = 64;
    constexpr int STAGES = 4;

    constexpr int NUM_WARPS = (CTA_M / WARP_M) * (CTA_N / WARP_N);
    constexpr int kSmemByteSize = (CTA_M * (CTA_K + SMEM_PAD_A) + CTA_N * (CTA_K + SMEM_PAD_B) / kInterleave + CTA_N) * STAGES * sizeof(half);
    if (kSmemByteSize >= 99 * 1024)
    {
      printf("This kernel requires %d Bytes of shared memory, which exceeds device limit.\n", kSmemByteSize);
      return _out_feats;
    }
    int j_factors1 = num_out_channels / CTA_N / 1;
    dim3 num_blocks((num_out_feats + CTA_M - 1) / CTA_M * j_factors1);
    dim3 threads_per_block(WARP_SIZE, NUM_WARPS);
    auto kernel_func = gemm_w4a16_T2<CTA_M, CTA_N, CTA_K, WARP_M, WARP_N, WARP_K, STAGES, G>;
    hipFuncSetAttribute(reinterpret_cast<const void*>(kernel_func), hipFuncAttributeMaxDynamicSharedMemorySize, kSmemByteSize);
    kernel_func<<<num_blocks, threads_per_block, kSmemByteSize>>>(
        in_feats, kernel, scales, zeros, out_feats, num_in_feats, num_out_channels, num_in_channels);
  }

  return _out_feats;
}