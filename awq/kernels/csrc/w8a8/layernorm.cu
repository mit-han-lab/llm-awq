#include "hip/hip_runtime.h"
// Inspired by QServe https://github.com/mit-han-lab/qserve/tree/main.
// Modified by Yuming Lou.
// @article{lin2024awq,
//   title={AWQ: Activation-aware Weight Quantization for On-Device LLM Compression and Acceleration},
//   author={Lin, Ji and Tang, Jiaming and Tang, Haotian and Yang, Shang and Chen, Wei-Ming and Wang, Wei-Chen and Xiao, Guangxuan and Dang, Xingyu and Gan, Chuang and Han, Song},
//   journal={Proceedings of Machine Learning and Systems},
//   volume={6},
//   pages={87--100},
//   year={2024}
// }
#include <ATen/cuda/HIPContext.h>
#include <torch/extension.h>
#include "dispatch_utils.h"
#include "utils.cuh"
#include "reduction_utils.cuh"


namespace vllm {

// from TRTLLM
template <typename Tf, typename T>
__inline__ __device__ Tf compute_layernorm(Tf val, float s_mean, float s_variance, const T* gamma, const T* beta, int i)
{
    Tf ret = (val - s_mean) * s_variance * cuda_cast<Tf>(gamma[i]);
    if (beta != nullptr)
    {
        ret = ret + cuda_cast<Tf>(beta[i]);
    }
    return ret;
}

// from TRTLLM
/* Computes the layernorm https://pytorch.org/docs/stable/generated/torch.nn.LayerNorm.html
 * normed_output <- ( (input - E[input]) / Sqrt(Var[input] + eps) ) * gamma + beta
 * input is [tokens, hidden_dim]. Mean and Variance are per-row (i.e. per-token)
 *
 * One CTA handles one row.
 *
 * with USE_DIFF_OF_SQUARES set to false:
 * First pass (loop) computes the mean.
 * Second computes the variance via Var[x] = E[(x - E[x])²].
 * Third pass computes and writes normed_output
 * For better speedup, we set USE_DIFF_OF_SQUARES to true (may be faster but less accurate):
 * It turns out the accuracy dosen't drop.
 * First pass (loop) computes the mean and variance via Var[x] = E[x²] - E[x]²
 * Second pass computes and writes normed_output
 * 
 *
 * use_shmem controls if we cache input values into shared memory
 *
 * Optional: with dynamic scaling, the last pass doesn't write immediately but finds the
 *           amax per row. A final pass scales to int8 accordingly, and writes output to
 *           normed_output_quant.
 */
template <typename T, typename scale_type, bool USE_DIFF_OF_SQUARES = true>
__global__ void generalLayerNorm(const T* input, const T* gamma, const T* beta, T* normed_output, const float eps,
    int tokens, int hidden_dim, const scale_type* scale_orig_quant_per_tensor, scale_type* scale_orig_quant_per_token,
    int8_t* normed_output_quant, bool use_shmem)
{
    constexpr auto num_elems_T = num_elems<T>::value;
    using int8_packed_t = typename packed_as<int8_t, num_elems_T>::type;
    using float_packed_t = typename packed_as<float, num_elems_T>::type;
    using T_scalar = typename packed_as<T, 1>::type;

    extern __shared__ __align__(sizeof(float)) char _shmem[];
    T* shmem = reinterpret_cast<T*>(_shmem);
    __shared__ float s_mean;
    __shared__ float s_variance;

    const int tidx = threadIdx.x;
    const int bidx = blockIdx.x;

    float mean = 0.0f;
    float variance = 0.0f;
    float local_sum = 0.0f;
    float local_var_sum = 0.0f;
    const int n_elems = hidden_dim / num_elems_T;
    for (int i = tidx; i < n_elems; i += blockDim.x)
    {
        const T val = input[bidx * n_elems + i];
        if (use_shmem)
        {
            shmem[i] = val;
        }
        const float_packed_t val_f = cuda_cast<float_packed_t>(val);
        local_sum += cuda_sum<float>(val_f);
        if (USE_DIFF_OF_SQUARES)
        {
            local_var_sum += cuda_sum<float>(val_f * val_f);
        }
    }
    //Compute mean
    if (USE_DIFF_OF_SQUARES)
    {
        float packed[2] = {local_sum, local_var_sum};
        blockReduceSumV2<float, 2>(packed);
        mean = packed[0];
        variance = packed[1];
    }
    else
    {
        mean = blockReduceSum(local_sum);
    }

    if (threadIdx.x == 0)
    {
        mean = mean / hidden_dim;
        s_mean = mean;
        if (USE_DIFF_OF_SQUARES)
        {
            variance = (variance / hidden_dim) - (mean * mean); // Var[x] = E[x²] - E[x]²
            s_variance = rsqrtf(variance + eps);
        }
    }
    __syncthreads();


    if (!USE_DIFF_OF_SQUARES)
    {
        for (int i = tidx; i < n_elems; i += blockDim.x)
        {
            const T val = use_shmem ? shmem[i] : input[bidx * n_elems + i];
            float_packed_t diff = cuda_cast<float_packed_t>(val); // - s_mean;
            local_var_sum += cuda_sum<float>(diff * diff);
        }
        variance = blockReduceSum(local_var_sum);

        if (threadIdx.x == 0)
        {
            s_variance = rsqrtf(variance / hidden_dim + eps);
        }
        __syncthreads();
    }

    // Compute LN and Quantize
    const bool with_per_token_scaling = scale_orig_quant_per_token != nullptr;
    const bool with_per_tensor_scaling = scale_orig_quant_per_tensor != nullptr;
    const float_packed_t scale_orig_quant
        = cuda_cast<float_packed_t>(with_per_tensor_scaling ? __half2float(*scale_orig_quant_per_tensor) : 0.0f);
    T_scalar amax = 1e-6f;

    for (int i = tidx; i < n_elems; i += blockDim.x)
    {
        const int index = bidx * n_elems + i;
        const float_packed_t val_f = cuda_cast<float_packed_t>(use_shmem ? shmem[i] : input[index]);
        const T val = cuda_cast<T>(compute_layernorm(val_f, s_mean, s_variance, gamma, beta, i));

        if (with_per_token_scaling)
        {
            amax = cuda_max(cuda_max<T_scalar, T>(cuda_abs(val)), amax);
            if (use_shmem)
            {
                shmem[i] = val;
            }
        }
        else if (with_per_tensor_scaling)
        {
            reinterpret_cast<int8_packed_t*>(normed_output_quant)[index]
                = cuda_cast<int8_packed_t>(cuda_cast<float_packed_t>(val) * scale_orig_quant);
        }
        else
        {
            normed_output[index] = val;
        }
    }

    if (with_per_token_scaling)
    {
        float abs_max_f = blockAllReduceMax(cuda_cast<float>(amax));
        const float dynamic_per_token_scale = 127.f / abs_max_f;
        for (int i = tidx; i < n_elems; i += blockDim.x)
        {
            const int index = bidx * n_elems + i;
            float_packed_t val_f = cuda_cast<float_packed_t>(use_shmem ? shmem[i] : input[index]);
            if (!use_shmem)
            {
                val_f = compute_layernorm(val_f, s_mean, s_variance, gamma, beta, i);
            }

            reinterpret_cast<int8_packed_t*>(normed_output_quant)[index]
                = cuda_cast<int8_packed_t>(val_f * cuda_cast<float_packed_t>(dynamic_per_token_scale));
        }
        if (tidx == 0)
        {
            scale_orig_quant_per_token[bidx] = abs_max_f / 127.f;
        }
    }
}


} // namespace vllm

void layer_norm_general(torch::Tensor &out,    // [..., hidden_size]
              torch::Tensor &input,  // [..., hidden_size]
              torch::Tensor &weight, // [hidden_size]
              torch::Tensor &bias, // [hidden_size]
              torch::Tensor &scaling, // [tokens] or [1]
              float epsilon,
              bool use_per_token_quant = true) {
  int hidden_size = input.size(-1);
  int num_tokens = input.numel() / hidden_size;
  dim3 grid(num_tokens);
  dim3 block(std::min(hidden_size, 128));//Reduce the idle probability of threads
  block.x = 32 * ((block.x + 31) / 32);
  
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  VLLM_DISPATCH_FLOATING_TYPES(input.scalar_type(), "generalLayerNorm", [&] {
    using T = typename FloatTypeConverter<scalar_t>::Type;
    if (use_per_token_quant) {
      // per-token
      vllm::generalLayerNorm<T, at::Half><<<grid, block, 0, stream>>>(
        reinterpret_cast<T*>(input.data_ptr<scalar_t>()), 
        reinterpret_cast<T*>(weight.data_ptr<scalar_t>()), 
        reinterpret_cast<T*>(bias.data_ptr<scalar_t>()),
        nullptr, epsilon, num_tokens, hidden_size, nullptr, scaling.data_ptr<at::Half>(),
        out.data_ptr<int8_t>(), false
      );
      // input, gamma, beta, normed_output, eps, tokens, hidden_dim, per_tensor_scale, per_token_scale
      // normed_output_quant, use_shmem
        // out.data_ptr<int8_t>(), input.data_ptr<scalar_t>(),
        // weight.data_ptr<scalar_t>(), epsilon, num_tokens, hidden_size);
    } else {
      // per-tensor
      vllm::generalLayerNorm<T, at::Half><<<grid, block, 0, stream>>>(
        reinterpret_cast<T*>(input.data_ptr<scalar_t>()), 
        reinterpret_cast<T*>(weight.data_ptr<scalar_t>()), nullptr,
        nullptr, epsilon, num_tokens, hidden_size, scaling.data_ptr<at::Half>(), nullptr,
        out.data_ptr<int8_t>(), false
      );
    }
  });
}